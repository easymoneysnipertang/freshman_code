#include "hip/hip_runtime.h"
﻿#include<iostream>
#include<fstream>
#include<string>
#include<vector>
#include<sstream>
#include<Windows.h>
#include<algorithm>
#include"hip/hip_runtime.h"
#include""
#include <thrust/device_vector.h>

using namespace std;

const int BLOCK_SIZE = 1024;

class InvertedIndex {// 倒排索引结构
public:
	int length = 0;
	vector<unsigned int> docIdList;
};
// 重载比较符，以长度排序各个索引
bool operator<(const InvertedIndex& i1, const InvertedIndex& i2) {
	return i1.length < i2.length;
}

// 把倒排列表按长度排序
void sorted(int* list, vector<InvertedIndex>& idx, int num) {
	for (int i = 0; i < num - 1; i++) {
		for (int j = 0; j < num - i - 1; j++) {
			if (idx[list[j]].length > idx[list[j + 1]].length) {
				int tmp = list[j];
				list[j] = list[j + 1];
				list[j + 1] = tmp;
			}
		}
	}
}
// svs实现
InvertedIndex SVS(int* queryList, vector<InvertedIndex>& index, int num) {
	InvertedIndex s = index[queryList[0]];// 取最短的列表

	// 与剩余列表求交
	for (int i = 1; i < num; i++) {
		int count = 0;// s从头往后遍历一遍
		int t = 0;
		// s列表中的每个元素都拿出来比较
		for (int j = 0; j < s.length; j++) {// 所有元素都得访问一遍
			bool isFind = false;// 标志，判断当前count位是否能求交

			for (; t < index[queryList[i]].length; t++) {
				// 遍历i列表中所有元素
				if (s.docIdList[j] == index[queryList[i]].docIdList[t]) {
					isFind = true;
					break;
				}
				else if (s.docIdList[j] < index[queryList[i]].docIdList[t])// 升序排列
					break;
			}
			if (isFind)// 覆盖
				s.docIdList[count++] = s.docIdList[j];
		}
		if (count < s.length)// 最后才做删除
			s.docIdList.erase(s.docIdList.begin() + count, s.docIdList.end());
		s.length = count;
	}
	return s;
}

// adp实现
class QueryItem {
public:
	int cursor;// 当前读到哪了
	int end;// 倒排索引总长度
	int key;// 关键字值
};
bool operator<(const QueryItem& q1, const QueryItem& q2) {// 选剩余元素最少的元素
	return (q1.end - q1.cursor) < (q2.end - q2.cursor);
}
InvertedIndex ADP(int* queryList, vector<InvertedIndex>& index, int num)
{
	InvertedIndex S;
	QueryItem* list = new QueryItem[num]();
	for (int i = 0; i < num; i++)// 预处理
	{
		list[i].cursor = 0;
		list[i].key = queryList[i];
		list[i].end = index[queryList[i]].docIdList.size();
	}
	for (int i = list[0].cursor; i < list[0].end; i++) {// 最短的列表非空
		bool isFind = true;
		unsigned int e = index[list[0].key].docIdList[i];
		for (int s = 1; s != num && isFind == true; s++) {
			isFind = false;
			while (list[s].cursor < list[s].end) {// 检查s列表
				if (e == index[list[s].key].docIdList[list[s].cursor]) {
					isFind = true;
					break;
				}
				else if (e < index[list[s].key].docIdList[list[s].cursor])
					break;
				list[s].cursor++;// 当前访问过，且没找到合适的，往后移
			}
			// 下一个链表
		}
		// 当前元素已被访问过
		if (isFind)
			S.docIdList.push_back(e);
		//sort(list, list + num);// 重排，将未探查元素少的列表前移
	}
	return S;
}


__global__ void cmp_kernel(int* index, int cursor,int e,bool* isFind) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;//计算线程索引
	if (e == index[cursor+tid])
		*isFind = true;
}


//返回第一个大于等于目标元素的下标
__device__ int find1stGreaterEqual(const int* arr, unsigned int target, int begin, int end) {
	int left = begin;
	int right = end;
	while (left < right)
	{
		int mid = left + (right - left) / 2;
		if (arr[mid] < target)
		{
			left = mid + 1;
		}
		else
			right = mid;
	}
	return left;
}
__global__ void ADP_kernel(int* index, int* lengthArr) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;//计算线程索引
	int totalThreads = gridDim.x * blockDim.x;

	int num = lengthArr[0];

	// 线程起始，结束
	int start = lengthArr[1] / totalThreads * tid, end = min(lengthArr[1] / totalThreads * (tid + 1), lengthArr[1]);// 第一个链表拿来做划分

	QueryItem* list = new QueryItem[num]();
	int count = lengthArr[1];
	for (int i = 1; i < num; i++)// 预处理
	{
		// 起始结束位置
		list[i].cursor = find1stGreaterEqual(index, index[start], count, count + lengthArr[i + 1]);
		list[i].end = count + lengthArr[i + 1];
		count += lengthArr[i + 1];
	}

	int position = start;
	for (int i = start; i < end; i++) {
		bool isFind = true;
		unsigned int e = index[i];
		for (int s = 1; s != num && isFind == true; s++) {
			isFind = false;
			while (list[s].cursor < list[s].end) {// 检查s列表
				if (e == index[list[s].cursor]) {
					isFind = true;
					break;
				}
				else if (e < index[list[s].cursor])
					break;
				list[s].cursor++;// 当前访问过，且没找到合适的，往后移
			}
			// 下一个链表
		}
		// 当前元素已被访问过
		if (isFind)
			index[position++] = e;
		// TODO：先不管，能跑起来再说

	}
	delete list;
}

int main() {
	// 读取二进制文件
	fstream file;
	file.open("ExpIndex", ios::binary | ios::in);
	if (!file.is_open()) {
		cout << "Wrong in opening file!";
		return;

	}
	static vector<InvertedIndex>* invertedLists = new vector<InvertedIndex>();

	for (int i = 0; i < 2000; i++)		//总共读取2000个倒排链表
	{
		InvertedIndex* t = new InvertedIndex();				//一个倒排链表
		file.read((char*)&t->length, sizeof(t->length));
		for (int j = 0; j < t->length; j++)
		{
			unsigned int docId;			//文件id
			file.read((char*)&docId, sizeof(docId));
			t->docIdList.push_back(docId);//加入至倒排表
		}
		sort(t->docIdList.begin(), t->docIdList.end());//对文档编号排序
		invertedLists->push_back(*t);		//加入一个倒排表
	}
	file.close();

	// 读取查询数据
	file.open("ExpQuery", ios::in);
	static int query[1000][5] = { 0 };// 单个查询最多5个docId,全部读取
	string line;
	int count = 0;

	while (getline(file, line)) {// 读取一行
		stringstream ss; // 字符串输入流
		ss << line; // 传入这一行
		int i = 0;
		int temp;
		ss >> temp;
		while (!ss.eof()) {
			query[count][i] = temp;
			i++;
			ss >> temp;
		}
		count++;// 总查询数
	}
	file.close();

	//------求交------
	cout << "------intersection begin-----" << endl;
	hipError_t ret;// 错误检查

	dim3 dimBlock(BLOCK_SIZE, 1);// 线程块
	dim3 dimGrid(1, 1);// 线程网格

	int* index;
	int* gpuIndex;

	double time = 0;

	for (int i = 0; i < count / 25; i++) {// count个查询
		int num = 0;// query查询项数
		for (int j = 0; j < 5; j++) {
			if (query[i][j] != 0) {
				num++;
			}
		}

		int* lengthArr = new int[num + 1];
		int* gpuLengthArr;
		ret = hipMalloc((void**)&gpuLengthArr, (num + 1) * sizeof(int));
		if (ret != hipSuccess) {
			fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(ret));
		}

		int totalLength = 0;
		// 获取各段长度
		lengthArr[0] = num;// 用0号位置来保存num
		for (int j = 0; j < num; j++) {
			int length = (*invertedLists)[query[i][j]].length;
			totalLength += length;
			lengthArr[j + 1] = length;
		}

		index = new int[totalLength];// 开辟一维数组
		ret = hipMalloc((void**)&gpuIndex, totalLength * sizeof(int));
		if (ret != hipSuccess) {
			fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(ret));
		}

		totalLength = 0;
		for (int j = 0; j < num; j++) {// 倒排链表全部放进一个二维数组->一维数组表示二维数组
			// 复制到数组当中去
			copy((*invertedLists)[query[i][j]].docIdList.begin(), (*invertedLists)[query[i][j]].docIdList.end(), index + totalLength);
			// 当前位置
			totalLength += lengthArr[j + 1];
		}

		hipEvent_t start, stop;// 计时器
		float elapsedTime = 0.0;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);// 开始计时

		// 传递到GPU端
		ret = hipMemcpy(gpuLengthArr, lengthArr, (num + 1) * sizeof(int), hipMemcpyHostToDevice);
		if (ret != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(ret));
		}
		// 复制数据到GPU端
		ret = hipMemcpy(gpuIndex, index, totalLength * sizeof(int), hipMemcpyHostToDevice);
		if (ret != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(ret));
		}

		// kernel！启动！
		ADP_kernel << <dimGrid, dimBlock >> > (gpuIndex, gpuLengthArr);
		hipDeviceSynchronize();

		hipError_t cudaStatus2 = hipGetLastError();
		if (cudaStatus2 != hipSuccess) {
			fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus2));
		}
		//cout << i << endl;

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);// 停止计时
		hipEventElapsedTime(&elapsedTime, start, stop);
		time += elapsedTime;

		//销毁计时器
		hipEventDestroy(start);
		hipEventDestroy(stop);

		delete index;
		delete lengthArr;
		hipFree(gpuIndex);
		hipFree(gpuLengthArr);
	}

	printf("GPU Time: %f ms\n", time);

	return 0;
}

//#include <stdio.h>
//
//// For the CUDA runtime routines (prefixed with "cuda_")
//#include <hip/hip_runtime.h>
//
//#include <hip/hip_runtime_api.h>
///**
// * CUDA Kernel Device code
// *
// * Computes the vector addition of A and B into C. The 3 vectors have the same
// * number of elements numElements.
// */
//__global__ void
//vectorAdd(const float* A, const float* B, float* C, int numElements)
//{
//    int i = blockDim.x * blockIdx.x + threadIdx.x;
//
//    if (i < numElements)
//    {
//        C[i] = A[i] + B[i];
//    }
//}
//
///**
// * Host main routine
// */
//int
//main(void)
//{
//    // Error code to check return values for CUDA calls
//    hipError_t err = hipSuccess;
//
//    // Print the vector length to be used, and compute its size
//    int numElements = 50000;
//    size_t size = numElements * sizeof(float);
//    printf("[Vector addition of %d elements]\n", numElements);
//
//    // Allocate the host input vector A
//    float* h_A = (float*)malloc(size);
//
//    // Allocate the host input vector B
//    float* h_B = (float*)malloc(size);
//
//    // Allocate the host output vector C
//    float* h_C = (float*)malloc(size);
//
//    // Verify that allocations succeeded
//    if (h_A == NULL || h_B == NULL || h_C == NULL)
//    {
//        fprintf(stderr, "Failed to allocate host vectors!\n");
//        exit(EXIT_FAILURE);
//    }
//
//    // Initialize the host input vectors
//    for (int i = 0; i < numElements; ++i)
//    {
//        h_A[i] = rand() / (float)RAND_MAX;
//        h_B[i] = rand() / (float)RAND_MAX;
//    }
//
//    // Allocate the device input vector A
//    float* d_A = NULL;
//    err = hipMalloc((void**)&d_A, size);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    // Allocate the device input vector B
//    float* d_B = NULL;
//    err = hipMalloc((void**)&d_B, size);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    // Allocate the device output vector C
//    float* d_C = NULL;
//    err = hipMalloc((void**)&d_C, size);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    // Copy the host input vectors A and B in host memory to the device input vectors in
//    // device memory
//    printf("Copy input data from the host memory to the CUDA device\n");
//    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    // Launch the Vector Add CUDA Kernel
//    int threadsPerBlock = 256;
//    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
//    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
//    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
//    err = hipGetLastError();
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    // Copy the device result vector in device memory to the host result vector
//    // in host memory.
//    printf("Copy output data from the CUDA device to the host memory\n");
//    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    // Verify that the result vector is correct
//    for (int i = 0; i < numElements; ++i)
//    {
//        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
//        {
//            fprintf(stderr, "Result verification failed at element %d!\n", i);
//            exit(EXIT_FAILURE);
//        }
//    }
//
//    printf("Test PASSED\n");
//
//    // Free device global memory
//    err = hipFree(d_A);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    err = hipFree(d_B);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    err = hipFree(d_C);
//
//    if (err != hipSuccess)
//    {
//        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
//        exit(EXIT_FAILURE);
//    }
//
//    // Free host memory
//    free(h_A);
//    free(h_B);
//    free(h_C);
//
//    printf("Done\n");
//    return 0;
//}
